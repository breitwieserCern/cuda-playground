/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// https://github.com/parallel-forall/code-samples/blob/master/posts/unified-memory/dataElem_um.cu


#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>

struct DataElement
{
  char *name;
  int value;
  float* data;
};

__global__
void Kernel(DataElement *elem) {
  printf("On device: name=%s, value=%d, threadidx=%d\n", elem->name, elem->value, threadIdx.x);

  elem->name[0] = 'd';
  elem->value++;
  elem->data[3+threadIdx.x] = 42;
}

void launch(DataElement *elem) {
  Kernel<<< 1, 2 >>>(elem);
  hipDeviceSynchronize();
}

int main(void)
{
  DataElement *e;
  hipMallocManaged((void**)&e, sizeof(DataElement));

  e->value = 10;
  hipMallocManaged((void**)&(e->name), sizeof(char) * (strlen("hello") + 1) );
  strcpy(e->name, "hello");
  hipMallocManaged((void**)&(e->data), sizeof(float) * 20 );

  launch(e);

  printf("On host: name=%s, value=%d, data[3]=%f, data[4]=%f\n", e->name, e->value, e->data[3], e->data[4]);

  hipFree(e->name);
  hipFree(e->data);
  hipFree(e);

  hipDeviceReset();
}
