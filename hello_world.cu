// https://devblogs.nvidia.com/parallelforall/even-easier-introduction-cuda/
// nvcc -o hello_world hello_world.cu
// nvprof ./hello_world


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// function to add the elements of two arrays
__global__
void add(int n, float *x, float *y) {
  // for (int i = 0; i < n; i++)
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

int main(void) {
  int N = 1<<20; // 1M elements

  // float *x = new float[N];
  // float *y = new float[N];

  // Allocate Unified Memory -- accessible from CPU or GPU
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the CPU
  // add(N, x, y);
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(N, x, y);
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  // delete [] x;
  // delete [] y;
  hipFree(x);
  hipFree(y);

  return 0;
}
